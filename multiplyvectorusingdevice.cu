#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>

__device__ int multiply(int* a, int* b){
    return (*a) * (*b);

}

__global__ void dotproductvector(int*a , int*b , int*c , int N){
    int i = blockIdx.x *blockDim.x + threadIdx.x;
    

    if (i<N){
        c[i] = multiply(a+i,b+i);
    }
}

int main(){
    int N = 10;
    int size = N * sizeof(int);
    int * a = new int[N];
    int *b = new int[N];
    int *c = new int[N];

    for(int i = 0;i<N;i++){
        a[i] = i+1;
    }

    for(int i = 0; i < N; i++){
    b[i] = 20 - i; 
}



    int *a_in, *b_in, *c_in;
    hipMalloc((void**)&a_in,size);
    hipMalloc((void**)&b_in,size);
    hipMalloc((void**)&c_in,size);

    hipMemcpy(a_in,a,size,hipMemcpyHostToDevice);
    hipMemcpy(b_in,b,size,hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blockGrid = (N + threadsPerBlock-1 )/ threadsPerBlock;

    dotproductvector<<<threadsPerBlock, blockGrid>>>(a_in,b_in,c_in,N);

    hipMemcpy(c,c_in,size,hipMemcpyDeviceToHost);

    std::cout << "multiplied array: ";
    for (int i = 0; i < N; i++) {
        std::cout << a[i]<<" * "<<b[i]<<" = "<<c[i] << " \n";
        }
    std::cout << std::endl;


    return 0;

}