#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const int *a, const int *b, int *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n){
         c[idx] = a[idx] + b[idx];
    }
       
}

int main(){
    int n = 10;
    size_t size = n * sizeof(int);

    // Allocate host (CPU) arrays
    int h_a[10], h_b[10], h_c[10];

    // Initialize the input arrays
    for (int i = 0; i < n; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    

    int *d_a, *d_b, *d_c;

    // Allocate device memory
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);


    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    hipDeviceSynchronize();

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
    }

    return 0;
}